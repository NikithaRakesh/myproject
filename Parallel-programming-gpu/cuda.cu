#include <iostream>
#include <fstream>
#include <string>
#include <map>
#include <vector>
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <sys/resource.h>

using namespace std;

long getMemoryUsage() {
    struct rusage resourceUsage;
    getrusage(RUSAGE_SELF, &resourceUsage);
    return resourceUsage.ru_maxrss; // Return maximum resident set size used (in kilobytes)
}

__global__ void calculateFrequenciesKernel(const char *sequence, int sequenceLength, int substringLength, char *subsequences, int totalSubsequences) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalSubsequences) {
        for (int j = 0; j < substringLength; ++j) {
            subsequences[idx * substringLength + j] = sequence[idx + j];
        }
    }
}

void calculateFrequenciesCUDA(const string &sequence, int substringLength, map<string, int> &frequencyMap, int blockSize) {
    int sequenceLength = sequence.size();
    int totalSubsequences = sequenceLength - substringLength + 1;
    

    char *d_sequence, *d_subsequences;
    hipMalloc((void **)&d_sequence, sequenceLength * sizeof(char));
    hipMalloc((void **)&d_subsequences, totalSubsequences * substringLength * sizeof(char));

    hipMemcpy(d_sequence, sequence.c_str(), sequenceLength * sizeof(char), hipMemcpyHostToDevice);


    int numBlocks = (totalSubsequences + blockSize - 1) / blockSize;

    calculateFrequenciesKernel<<<numBlocks, blockSize>>>(d_sequence, sequenceLength, substringLength, d_subsequences, totalSubsequences);

    char *subsequences = new char[totalSubsequences * substringLength];
    hipMemcpy(subsequences, d_subsequences, totalSubsequences * substringLength * sizeof(char), hipMemcpyDeviceToHost);

    for (int i = 0; i < totalSubsequences; ++i) {
        string subsequence(subsequences + i * substringLength, substringLength);
        ++frequencyMap[subsequence];
    }

    hipFree(d_sequence);
    hipFree(d_subsequences);
    delete[] subsequences;
}

int main(int argc, char *argv[]) {
    long memoryAtStart = getMemoryUsage();
    auto startTime = chrono::high_resolution_clock::now();

    char *maxProtLenEnv = getenv("MAXPROTLEN");
    char *cpuCoresEnv = getenv("MAXCORES");

    int maxProteinLength = maxProtLenEnv ? stoi(maxProtLenEnv) : 1;
    int cpuCores = cpuCoresEnv ? stoi(cpuCoresEnv) : 1;

    string sequence;
    ifstream fastaFile("MusMusculus.fasta");
    if (!fastaFile.is_open()) {
        cerr << "Error: Unable to open the FASTA file." << endl;
        return 1;
    }

    string line;
    while (getline(fastaFile, line)) {
        if (line.empty() || line[0] == '>') continue;
        sequence += line;
    }
    fastaFile.close();

    map<string, int> frequencyMap;
    calculateFrequenciesCUDA(sequence, maxProteinLength, frequencyMap, cpuCores);

    ofstream outputFile("output2048.csv");
    if (outputFile.is_open()) {
        for (const auto &pair : frequencyMap) {
            outputFile << pair.first << "," << pair.second << "\n";
        }
        outputFile.close();
    } else {
        cerr << "Error: Unable to open the output CSV file for writing." << endl;
        return 1;
    }

    auto endTime = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsedTime = endTime - startTime;
    long memoryUsed = getMemoryUsage() - memoryAtStart;

    cout << "Elapsed time: " << elapsedTime.count() << " seconds" << endl;
    cout << "Memory used: " << memoryUsed << " KB" << endl;
    cout << "CPU cores Used for calculating subsequence: " << cpuCores << endl;

    return 0;
}
